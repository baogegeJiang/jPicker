#include "hip/hip_runtime.h"
#include "mex.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>
#include <math.h>
using namespace std;
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}
__global__ void double2float(double *ain,float *aout,int len){
    int i=threadIdx.x+blockIdx.x*blockDim.x;
    while(i<len){
        aout[i]=(float)ain[i];
        i+=blockDim.x*gridDim.x;
    }
    
}

__global__ void corr( float * a,float* b,  int la,int lb,float tb0,double* c) {
    int i=threadIdx.x+blockIdx.x*blockDim.x,j=0;
    float ta=0,tc=0;
    
    while (i < la-lb+1) {
        ta=0;tc=0;
        for(j=0;j<lb;j++){
            ta+=a[i+j]*a[i+j];
            tc+=a[i+j]*b[j];
        }
        
        *(c+i)=(double) tc/(sqrtf(ta)*tb0);
        if(ta==0){
           *(c+i)=0;
        }
        i+=blockDim.x*gridDim.x;
    }
}
void mexFunction(int nlhs,mxArray *plhs[],int nrhs,const mxArray *prhs[])
{
    double *a,*b,*c;
    float tb0f=0;
    a = mxGetPr(prhs[0]);
    b = mxGetPr(prhs[1]);
    const int la=*(mxGetPr(prhs[2]));
    const int lb=*(mxGetPr(prhs[3]));
    int blocksPerGrid = 256;
    int threadsPerBlock = 256;
    double tb=0;
    plhs[0]=mxCreateDoubleMatrix(la-lb+1,1,mxREAL);
    c= mxGetPr(plhs[0]);
    
    for(int i=0;i<lb;i++){
        tb+=b[i]*b[i];
    }
    tb=sqrt(tb);tb0f=(float)tb;
    
    float * d_a, *d_b;
    double * dd_a, *dd_b,*d_c;
    hipMalloc(&dd_a, sizeof(double) * la);
    hipMalloc(&dd_b, sizeof(double) * lb);
    hipMalloc(&d_a, sizeof(float) * la);
    hipMalloc(&d_b, sizeof(float) * lb);
    hipMalloc(&d_c, sizeof(double) * (la-lb+1));
    
    hipMemcpy(dd_a, a, sizeof(double) * la, hipMemcpyHostToDevice);
    hipMemcpy(dd_b, b, sizeof(double) * lb, hipMemcpyHostToDevice);
    double2float<<<blocksPerGrid, threadsPerBlock>>>(dd_a,d_a,la);
    double2float<<<blocksPerGrid, threadsPerBlock>>>(dd_b,d_b,lb);
    checkCUDAError("memcpy");
    
    
    corr<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b,la,lb,tb0f,d_c);
    checkCUDAError("corr fail");
    hipMemcpy(c, d_c, sizeof(double) * (la-lb+1), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(dd_a);
    hipFree(dd_b);
    
    return;
}
