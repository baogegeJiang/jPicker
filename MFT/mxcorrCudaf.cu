#include "hip/hip_runtime.h"
#include "mex.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>
#include <math.h>
using namespace std;
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

__global__ void corr( float * a,float* b,  int la,int lb,float tb0,float* c) {
    int i=threadIdx.x+blockIdx.x*blockDim.x,j=0;
    float ta=0,tc=0;
   
    while (i < la-lb+1) {
        ta=0;tc=0;
        for(j=0;j<lb;j++){
          ta+=a[i+j]*a[i+j];
          tc+=a[i+j]*b[j];
        }
       *(c+i)=tc/(sqrtf(ta)*tb0);
          i+=blockDim.x*gridDim.x;
    }
}
void mexFunction(int nlhs,mxArray *plhs[],int nrhs,const mxArray *prhs[])
{
    double *a,*b,*c;
    a = mxGetPr(prhs[0]);
    b = mxGetPr(prhs[1]);
    const int la=*(mxGetPr(prhs[2]));
    const int lb=*(mxGetPr(prhs[3]));
    float tb=0;
    float *af,*bf,*cf;
    af=new float[la];
    bf=new float[lb];
    cf=new float[la-lb+1];
    for (int i=0;i<la;i++){
        af[i]=(float) a[i];}
    for (int i=0;i<lb;i++){
        bf[i]=(float) b[i];}
    plhs[0]=mxCreateDoubleMatrix(la-lb+1,1,mxREAL);
    c= mxGetPr(plhs[0]);
    float* d_a, *d_b, *d_c;
    for(int i=0;i<lb;i++){
       tb+=bf[i]*bf[i];
    }
    tb=(float)sqrt((double)tb);
    hipMalloc(&d_a, sizeof(float) * la);
    hipMalloc(&d_b, sizeof(float) * lb);
    hipMalloc(&d_c, sizeof(float) * (la-lb+1));
    hipMemcpy(d_a, af, sizeof(float) * la, hipMemcpyHostToDevice);
    hipMemcpy(d_b, bf, sizeof(float) * lb, hipMemcpyHostToDevice);
    checkCUDAError("memcpy");

    int blocksPerGrid = 256;
    int threadsPerBlock = 256;
    corr<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b,la,lb,tb,d_c);
    checkCUDAError("corr fail");
    hipMemcpy(cf, d_c, sizeof(float) * (la-lb+1), hipMemcpyDeviceToHost);
    for(int i=0;i<la-lb+1;i++){
        c[i]=(double)cf[i];}
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete [] af;
    delete [] bf;
    delete [] cf;
    return;
}
