#include "hip/hip_runtime.h"
#include "mex.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>
#include <math.h>
using namespace std;
//plhs[0] = mxCreateNumericMatrix(2, 3, mxSINGLE_CLASS, mxREAL);
// fill in plhs[0] to contain the same as single([1 2 3; 4 5 6]); 
//  float * data = (float *) mxGetData(plhs[0]);
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}


__global__ void corr( float * a,float* b,  int la,int lb,float tb0,float* c) {
    int i=threadIdx.x+blockIdx.x*blockDim.x,j=0;
    float ta=0,tc=0;
    
    while (i < la-lb+1) {
        ta=0;tc=0;
        for(j=0;j<lb;j++){
            ta+=a[i+j]*a[i+j];
            tc+=a[i+j]*b[j];
        }
        
        *(c+i)= tc/(sqrtf(ta)*tb0);
        if(ta==0){
           *(c+i)=0;
        }
        i+=blockDim.x*gridDim.x;
    }
}
void mexFunction(int nlhs,mxArray *plhs[],int nrhs,const mxArray *prhs[])
{
    float *a,*b,*c;
    a = (float *) mxGetPr(prhs[0]);
    b = (float *) mxGetPr(prhs[1]);
    const int la=*(mxGetPr(prhs[2]));
    const int lb=*(mxGetPr(prhs[3]));
    int blocksPerGrid = 256;
    int threadsPerBlock = 256;
    plhs[0]=mxCreateNumericMatrix(la-lb+1,1, mxSINGLE_CLASS, mxREAL);
    c= (float *) mxGetPr(plhs[0]);
    float tb=0;
    for(int i=0;i<lb;i++){
        tb+=b[i]*b[i];
    }
    tb=sqrt(tb);
    
    float * d_a, *d_b,*d_c;
    hipMalloc(&d_a, sizeof(float) * la);
    hipMalloc(&d_b, sizeof(float) * lb);
    hipMalloc(&d_c, sizeof(float) * (la-lb+1));
    
    hipMemcpy(d_a, a, sizeof(float) * la, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * lb, hipMemcpyHostToDevice); 
    corr<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b,la,lb,tb,d_c);
    checkCUDAError("corr fail");
    hipMemcpy(c, d_c, sizeof(float) * (la-lb+1), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return;
}
