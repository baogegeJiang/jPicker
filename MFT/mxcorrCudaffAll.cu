#include "hip/hip_runtime.h"
#include "mex.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>
#include <math.h>
using namespace std;
//plhs[0] = mxCreateNumericMatrix(2, 3, mxSINGLE_CLASS, mxREAL);
// fill in plhs[0] to contain the same as single([1 2 3; 4 5 6]); 
//  float * data = (float *) mxGetData(plhs[0]);
const    int blocksPerGrid = 256;
const    int threadsPerBlock = 256;

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}


__global__ void corr( float * a,float* b,  int la,int lb,float tb0,float* c,float *m,float *s,float *mb,float *sb) {
    int i=threadIdx.x+blockIdx.x*blockDim.x,j=0;
    int tid=threadIdx.x;
    float ta=0,tc=0;
    mb=mb+blockIdx.x*blockDim.x;
    sb=sb+blockIdx.x*blockDim.x;
    mb[tid]=0;sb[tid]=0;
    while (i < la-lb+1) {
        ta=0;tc=0;
        for(j=0;j<lb;j++){
            ta+=a[i+j]*a[i+j];
            tc+=a[i+j]*b[j];
        }
        
        *(c+i)= tc/(sqrtf(ta)*tb0);
        if(ta==0){
           *(c+i)=0;
        }
        mb[tid]+=c[i];
        sb[tid]+=c[i]*c[i];
        i+=blockDim.x*gridDim.x;
    }
   __syncthreads();
   i=gridDim.x/2;
   while(i!=0){
     if(tid<i){
       mb[tid]+=mb[tid+i];//m[tid]/=2;
       sb[tid]+=sb[tid+i];//s[tid]/=2;
     }
     __syncthreads();
     i/=2;
     }    
      __syncthreads();
    if(tid==0){
     m[blockIdx.x]=mb[0];
     s[blockIdx.x]=sb[0];
     }
    return ;
}
void mexFunction(int nlhs,mxArray *plhs[],int nrhs,const mxArray *prhs[])
{
    float *a,*b,*c,*m,*s,mtmp[blocksPerGrid]={0},stmp[blocksPerGrid]={0};
    a = (float *) mxGetPr(prhs[0]);
    b = (float *) mxGetPr(prhs[1]);
    int la=*(mxGetPr(prhs[2]));
    int lb=*(mxGetPr(prhs[3]));
   
    plhs[0]=mxCreateNumericMatrix(la-lb+1,1, mxSINGLE_CLASS, mxREAL);
    c= (float *) mxGetPr(plhs[0]);
    float tb=0;
    for(int i=0;i<lb;i++){
        tb+=b[i]*b[i];
    }
    tb=sqrt(tb);
    
    float * d_a, *d_b,*d_c,*d_m,*d_s,*b_m,*b_s;
    hipMalloc(&d_a, sizeof(float) * la);
    hipMalloc(&d_b, sizeof(float) * lb);
    hipMalloc(&d_c, sizeof(float) * (la-lb+1));
    hipMalloc(&d_m, sizeof(float) *blocksPerGrid);    
    hipMalloc(&d_s, sizeof(float) *blocksPerGrid);
    hipMalloc(&b_m, sizeof(float) *blocksPerGrid*threadsPerBlock);
    hipMalloc(&b_s, sizeof(float) *blocksPerGrid*threadsPerBlock);
   // printf("%f %f\n",mtmp[0],stmp[0]);
   
    hipMemcpy(d_a, a, sizeof(float) * la, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * lb, hipMemcpyHostToDevice); 
    corr<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b,la,lb,tb,d_c,d_m,d_s,b_m,b_s);
    checkCUDAError("corr fail");
    hipMemcpy(c, d_c, sizeof(float) * (la-lb+1), hipMemcpyDeviceToHost);
    hipMemcpy(mtmp, d_m, sizeof(float) *blocksPerGrid, hipMemcpyDeviceToHost);
    hipMemcpy(stmp, d_s, sizeof(float) *blocksPerGrid, hipMemcpyDeviceToHost); 
    for(int i=1;i<blocksPerGrid;i++){
    mtmp[0]+=mtmp[i];stmp[0]+=stmp[i];
    }
    mtmp[0]/=(float)(la-lb+1);
    stmp[0]/=(float)(la-lb+1);
    stmp[0]=sqrt(stmp[0]-mtmp[0]*mtmp[0]);
     if( nlhs>=3){
      plhs[1]=mxCreateNumericMatrix(1,1, mxSINGLE_CLASS, mxREAL);
      plhs[2]=mxCreateNumericMatrix(1,1, mxSINGLE_CLASS, mxREAL);
      m= (float *) mxGetPr(plhs[1]);s= (float *) mxGetPr(plhs[2]);
      m[0]=mtmp[0];s[0]=stmp[0];
      la=la-lb+1;a=c;
      lb=*(mxGetPr(prhs[4]));
      float  mul=*(float *)(mxGetPr(prhs[5]));
      float tmin=m[0]+s[0]*mul;
      plhs[3]=mxCreateNumericMatrix(la-lb+1,1, mxSINGLE_CLASS, mxREAL);
      c= (float *)mxGetPr(plhs[3]);
      for( int i=0;i<la;i++)
      {
        if (i<la-lb+1) c[i]=a[i];
        if(a[i]>=tmin){
            for(int j=1;j<lb&&i-j>0&&i-j<la-lb+1;j++){
                if(a[i]>c[i-j]) c[i-j]=a[i];
          }

       }
     }


    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_m);
    hipFree(d_s);
    hipFree(b_m);
    hipFree(b_s);
    return;
}
