#include "hip/hip_runtime.h"
#include "mex.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>
#include <math.h>
using namespace std;
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

__global__ void corr( double* a,double* b,  int la,int lb,double tb0,double* c) {
    int i=threadIdx.x+blockIdx.x*blockDim.x,j=0;
    double ta=0,tc=0;
   
    while (i < la-lb+1) {
        ta=0;tc=0;
        for(j=0;j<lb;j++){
          ta+=a[i+j]*a[i+j];
          tc+=a[i+j]*b[j];
        }
       *(c+i)=tc/(sqrt(ta)*tb0);
          i+=blockDim.x*gridDim.x;
    }
}
void mexFunction(int nlhs,mxArray *plhs[],int nrhs,const mxArray *prhs[])
{
    double *a,*b,*c;
    //double t0,t1;
   // int la,lb;
    a = mxGetPr(prhs[0]);
    b = mxGetPr(prhs[1]);
    const int la=*(mxGetPr(prhs[2]));
    const int lb=*(mxGetPr(prhs[3]));
   // double *ta=new double(la);
  // double *tc=new double(la);
    double tb=0;
    plhs[0]=mxCreateDoubleMatrix(la-lb+1,1,mxREAL);
    c= mxGetPr(plhs[0]);
    double* d_a, *d_b, *d_c;
    for(int i=0;i<lb;i++){
       tb+=b[i]*b[i];
    }
    tb=sqrt(tb);
    //t0=(double)clock();
    hipMalloc(&d_a, sizeof(double) * la);
    hipMalloc(&d_b, sizeof(double) * lb);
    hipMalloc(&d_c, sizeof(double) * (la-lb+1));
    hipMemcpy(d_a, a, sizeof(double) * la, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(double) * lb, hipMemcpyHostToDevice);
    checkCUDAError("memcpy");
    //t1=(double)clock();
    //cout<< "mem assign "<<(t1-t0)/CLOCKS_PER_SEC<<endl;


    int blocksPerGrid = 256;
    int threadsPerBlock = 256;
    //t0=(double)clock();
    corr<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b,la,lb,tb,d_c);
    checkCUDAError("corr fail");
     //t1=(double)clock();
    //cout<< "cal assign "<<(t1-t0)/CLOCKS_PER_SEC<<endl;
    //t0=(double)clock();
    hipMemcpy(c, d_c, sizeof(double) * (la-lb+1), hipMemcpyDeviceToHost);
    //t1=(double)clock();
    //cout<< "mem back "<<(t1-t0)/CLOCKS_PER_SEC<<endl;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return;
}