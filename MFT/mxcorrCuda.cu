#include "hip/hip_runtime.h"
#include "mex.h"
#include <cstdio>
#include <hip/hip_runtime.h>
using namespace std;
void checkCUDAError(const char *msg)   
{   
    hipError_t err = hipGetLastError();   
    if( hipSuccess != err)    
    {   
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,    
                                  hipGetErrorString( err) );   
        exit(EXIT_FAILURE);   
    }                            
}  

__global__ void corr( double* a, double* b,   int la,int lb,double tb,double* c) {
    int idx = threadIdx.x+blockIdx.x*blockDim.x;
    int i=idx,j=0;
    double ta=0,tc=0;
    double tb0=sqrt(tb);
    while (i < la-lb+1) {
        ta=0;tc=0;
        for(j=0;j<lb;j++){
          ta+=a[i+j]*a[i+j];
          tc+=a[i+j]*b[j];
        }
       *(c+i)=tc/(sqrt(ta)*tb0);
          i+=blockDim.x*gridDim.x;
    }
}
void mexFunction(int nlhs,mxArray *plhs[],int nrhs,const mxArray *prhs[])
{
    double *a,*b,*c;
    int la,lb;
    a = mxGetPr(prhs[0]);
    b = mxGetPr(prhs[1]);
    la=*(mxGetPr(prhs[2]));
    lb=*(mxGetPr(prhs[3]));

    double tb=0;
    plhs[0]=mxCreateDoubleMatrix(la-lb+1,1,mxREAL);
    c= mxGetPr(plhs[0]);
    double* d_a, *d_b, *d_c;
    for(int i=0;i<lb;i++){
       tb+=b[i]*b[i];
    }
    hipMalloc(&d_a, sizeof(double) * la);
    hipMalloc(&d_b, sizeof(double) * lb);
    hipMalloc(&d_c, sizeof(double) * (la-lb+1));
    hipMemcpy(d_a, a, sizeof(double) * la, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(double) * lb, hipMemcpyHostToDevice);
    checkCUDAError("memcpy");

    int blocksPerGrid = 256;
    int threadsPerBlock = 256;
    corr<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b,la,lb,tb,d_c);
    checkCUDAError("corr fail");
    hipMemcpy(c, d_c, sizeof(double) * (la-lb+1), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return;
}
