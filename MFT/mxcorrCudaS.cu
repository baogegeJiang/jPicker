#include "hip/hip_runtime.h"
#include "mex.h"
#include <cstdio>
#include <hip/hip_runtime.h>
using namespace std;
__constant__ double d_b[1000];
void checkCUDAError(const char *msg)   
{   
    hipError_t err = hipGetLastError();   
    if( hipSuccess != err)    
    {   
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,    
                                  hipGetErrorString( err) );   
        exit(EXIT_FAILURE);   
    }                            
}  

// Kernel 函数
__global__ void corr( double* a,int la,int lb,double tb,double* c) {
    int idx = threadIdx.x+blockIdx.x*blockDim.x;
    c[1]=1;
   // printf("%d",idx);
    int i=idx,j=0;
    double ta=0,tc=0;
    double tb0=sqrt(tb);
    while (i < la-lb+1) {
        ta=0;tc=0;
        for(j=0;j<lb;j++){
          ta+=a[i+j]*a[i+j];
          tc+=a[i+j]*d_b[j];
        }
       *(c+i)=tc/(sqrt(ta)*tb0);
          i+=blockDim.x*gridDim.x;
    }
}
void mexFunction(int nlhs,mxArray *plhs[],int nrhs,const mxArray *prhs[])
{
    double *a,*b,*c;
    int la,lb;
    a = mxGetPr(prhs[0]);
    b = mxGetPr(prhs[1]);
    la=*(mxGetPr(prhs[2]));
    lb=*(mxGetPr(prhs[3]));
   // double *ta=new double(la);
  // double *tc=new double(la);
    double tb=0;
    plhs[0]=mxCreateDoubleMatrix(la-lb+1,1,mxREAL);
    c= mxGetPr(plhs[0]);
    double* d_a, *d_c;
    for(int i=0;i<lb;i++){
       tb+=b[i]*b[i];
    }
    hipMalloc(&d_a, sizeof(double) * la);
//    hipMalloc(&d_b, sizeof(double) * lb);
    hipMalloc(&d_c, sizeof(double) * (la-lb+1));
    hipMemcpy(d_a, a, sizeof(double) * la, hipMemcpyHostToDevice);
//    hipMemcpy(d_b, b, sizeof(double) * lb, hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL(d_b),b,sizeof(double) * lb);
    checkCUDAError("memcpy");
    // 执行Kernel
    int blocksPerGrid = 256;
    int threadsPerBlock = 256;
    corr<<<blocksPerGrid, threadsPerBlock>>>(d_a,la,lb,tb,d_c);
    checkCUDAError("corr fail");
    hipMemcpy(c, d_c, sizeof(double) * (la-lb+1), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return;
}
