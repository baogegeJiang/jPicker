#include "hip/hip_runtime.h"
#include "mex.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>
#include <math.h>
using namespace std;
//plhs[0] = mxCreateNumericMatrix(2, 3, mxSINGLE_CLASS, mxREAL);
// fill in plhs[0] to contain the same as single([1 2 3; 4 5 6]); 
//  float * data = (float *) mxGetData(plhs[0]);
const    int blocksPerGrid = 256;
const    int threadsPerBlock = 256;

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}


__global__ void corr( float * a,float* b,  int la,int lb,float tb0,float* c,float *m,float *s,float *mb,float *sb) {
    int i=threadIdx.x+blockIdx.x*blockDim.x,j=0;
    int tid=threadIdx.x;
    float ta=0,tc=0;
    mb=mb+blockIdx.x*blockDim.x;
    sb=sb+blockIdx.x*blockDim.x;
    mb[tid]=0;sb[tid]=0;
    while (i < la-lb+1) {
        ta=0;tc=0;
        for(j=0;j<lb;j++){
            ta+=a[i+j]*a[i+j];
            tc+=a[i+j]*b[j];
        }
        
        *(c+i)= tc/(sqrtf(ta)*tb0);
        if(ta==0){
           *(c+i)=0;
        }
        mb[tid]+=c[i];
        sb[tid]+=c[i]*c[i];
        i+=blockDim.x*gridDim.x;
    }
   __syncthreads();
   i=gridDim.x/2;
   while(i!=0){
     if(tid<i){
       mb[tid]+=mb[tid+i];//m[tid]/=2;
       sb[tid]+=sb[tid+i];//s[tid]/=2;
     }
     __syncthreads();
     i/=2;
     }    
      __syncthreads();
    if(tid==0){
     m[blockIdx.x]=mb[0];
     s[blockIdx.x]=sb[0];
     }
    return ;
}
void mexFunction(int nlhs,mxArray *plhs[],int nrhs,const mxArray *prhs[])
{
    float *a0,*b0,*c0,*m,*s,mtmp[blocksPerGrid]={0},stmp[blocksPerGrid]={0},*addMat;
    a0 = (float *) mxGetPr(prhs[0]);
    b0 = (float *) mxGetPr(prhs[1]);
    int la0=*(mxGetPr(prhs[2]));
    int lb0=*(mxGetPr(prhs[3]));
    int lc0=*(mxGetPr(prhs[4]));
    int mul=*(float *)(mxGetPr(prhs[5]));
    int laout=*(mxGetPr(prhs[6]));
    int N=*(mxGetPr(prhs[7]));
    double *bIndex=mxGetPr(prhs[8]);
    plhs[0]=mxCreateNumericMatrix(laout,N, mxSINGLE_CLASS, mxREAL);
    plhs[1]=mxCreateNumericMatrix(1,N, mxSINGLE_CLASS, mxREAL);
    plhs[2]=mxCreateNumericMatrix(1,N, mxSINGLE_CLASS, mxREAL);
    plhs[3]=mxCreateNumericMatrix(laout,1, mxSINGLE_CLASS, mxREAL);
    float *addMatTmp=(float *)malloc(sizeof(float)*laout);
    m= (float *) mxGetPr(plhs[1]);s= (float *) mxGetPr(plhs[2]);
    addMat=(float *) mxGetPr(plhs[3]);
    c0= (float *) mxGetPr(plhs[0]);
    for(int i=0;i<laout;i++){
       addMat[i]=0;}
    for(int i=0;i<N;i++){
       for(int j=0;j<laout;j++){
          c0[i*laout+j]=0;
       }
    }
   
    float * d_a, *d_b,*d_c,*d_m,*d_s,*b_m,*b_s;

    hipMalloc(&d_a, sizeof(float) * la0);
    hipMalloc(&d_b, sizeof(float) * lb0);
    hipMalloc(&d_c, sizeof(float) * (la0-lb0+1));
    hipMalloc(&d_m, sizeof(float) *blocksPerGrid);    
    hipMalloc(&d_s, sizeof(float) *blocksPerGrid);
    hipMalloc(&b_m, sizeof(float) *blocksPerGrid*threadsPerBlock);
    hipMalloc(&b_s, sizeof(float) *blocksPerGrid*threadsPerBlock);
   
    float *a,*b,*c;
    int la,lb;
    float tb=0,tmin;
    for (int index=0;index<10;index++){
        la=la0;lb=lb0;
        int bI=(int)-bIndex[index]+1; 
        if(bI<0) continue;
        a=a0+index*la;
        b=b0+index*lb;
        c=c0+index*laout;
        tb=0;
        //continue;
        for(int i=0;i<lb;i++){
           tb+=b[i]*b[i];
         }
         tb=sqrt(tb); 
        hipMemcpy(d_a, a, sizeof(float) * la, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, sizeof(float) * lb, hipMemcpyHostToDevice); 
        //continue;
        corr<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b,la,lb,tb,d_c,d_m,d_s,b_m,b_s);
        //continue;
        checkCUDAError("corr fail");
        //printf("%d %d\n",(int)(c-c0),bI);
        hipMemcpy(c, d_c+bI, sizeof(float) * (la-lb+1-bI), hipMemcpyDeviceToHost);
        hipMemcpy(mtmp, d_m, sizeof(float) *blocksPerGrid, hipMemcpyDeviceToHost);
        hipMemcpy(stmp, d_s, sizeof(float) *blocksPerGrid, hipMemcpyDeviceToHost); 
        //continue;
        for(int i=1;i<blocksPerGrid;i++){
           mtmp[0]+=mtmp[i];stmp[0]+=stmp[i];
        }
        mtmp[0]/=(float)(la-lb+1);
        stmp[0]/=(float)(la-lb+1);
        stmp[0]=sqrt(stmp[0]-mtmp[0]*mtmp[0]);
        m[index]=mtmp[0];s[index]=stmp[0];
        la=la-lb+1;a=c;
        lb=lc0;
        tmin=m[index]+s[index]*mul;
        c= addMatTmp;
        for( int i=0;i<la;i++)
        {
            if (i<la-lb+1) c[i]=a[i];
            if(a[i]>=tmin){
                for(int j=1;j<lb&&i-j>0&&i-j<la-lb+1;j++){
                if(a[i]>c[i-j]) c[i-j]=a[i];
             }

             }
         }
        for(int i=0;i<la-lb+1;i++) addMat[i]+=c[i];

    }
    for(int i=0;i<laout;i++)
       addMat[i]/=(float)N;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_m);
    hipFree(d_s);
    hipFree(b_m);
    hipFree(b_s);
    free(addMatTmp);
    return;
}
