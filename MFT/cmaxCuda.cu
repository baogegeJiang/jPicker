#include "hip/hip_runtime.h"
#include "mex.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>
#include <math.h>
using namespace std;
//plhs[0] = mxCreateNumericMatrix(2, 3, mxSINGLE_CLASS, mxREAL);
// fill in plhs[0] to contain the same as single([1 2 3; 4 5 6]); 
//  float * data = (float *) mxGetData(plhs[0]);
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}


__global__ void cmax( float * a,int la,int lb,float* c,int loopN) {
    int si=(threadIdx.x+blockIdx.x*blockDim.x)*loopN;
    if(si>=la-lb+1) return;
    int i=si,j=0,ei=si+loopN; 
    float tmin; 
    for( ;i<la-lb+1&&i<ei;i++)
    {
      

        if(i>si)
        {
          if(tmin != a[i-1])
          {
              if(tmin> a[i+lb-1]) c[i]=tmin;
              else {c[i]=a[i+lb-1];tmin=c[i];}
              continue;
          }
        }
       
        tmin=-999999;
        for (j=0;j<lb;j++)
        {

            if(a[i+j]>tmin) tmin=a[i+j];
        }
        *(c+i)=tmin;
        }
}
void mexFunction(int nlhs,mxArray *plhs[],int nrhs,const mxArray *prhs[])
{
    float *a,*c;
    a = (float *) mxGetPr(prhs[0]);
    const int la=*(mxGetPr(prhs[1]));
    const int lb=*(mxGetPr(prhs[2]));
    int blocksPerGrid = 256;
    int threadsPerBlock = 256;
    plhs[0]=mxCreateNumericMatrix(la-lb+1,1, mxSINGLE_CLASS, mxREAL);
    c= (float *) mxGetPr(plhs[0]);
    
    float * d_a,*d_c;
    hipMalloc(&d_a, sizeof(float) * la);
    hipMalloc(&d_c, sizeof(float) * (la-lb+1));
    
    hipMemcpy(d_a, a, sizeof(float) * la, hipMemcpyHostToDevice);
    int loopN=(la-lb+1)/(blocksPerGrid*threadsPerBlock)+1;
  //  printf("%d",loopN);
   // return;
    cmax<<<blocksPerGrid, threadsPerBlock>>>(d_a,la,lb,d_c,loopN);
    checkCUDAError("corr fail");
    hipMemcpy(c, d_c, sizeof(float) * (la-lb+1), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_c);
    return;
}
