#include "hip/hip_runtime.h"
#include "mex.h"
using namespace std;
#include <cstdio>
#include <hip/hip_runtime.h>
using namespace std;
void checkCUDAError(const char *msg)   
{   
    hipError_t err = hipGetLastError();   
    if( hipSuccess != err)    
    {   
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,    
                                  hipGetErrorString( err) );   
        exit(EXIT_FAILURE);   
    }                            
}  

// Kernel 函数
__global__ void cmax( double* a, int la,int lb,double* c) {
    int idx = threadIdx.x+blockIdx.x*blockDim.x;
    c[1]=1;
   // printf("%d",idx);
    int i=idx,j=0;
    double tmpMax;
    while (i < la-lb+1) {
        tmpMax=a[i];
        for(j=1;j<lb;j++){
           tmpMax=max(tmpMax,*(a+i+j));
        }
       *(c+i)=tmpMax;
       i+=blockDim.x*gridDim.x;
    }
}
void mexFunction(int nlhs,mxArray *plhs[],int nrhs,const mxArray *prhs[])
{
    double *a,*c;
    int la,lb;
    a = mxGetPr(prhs[0]);
    la=*(mxGetPr(prhs[1]));
    lb=*(mxGetPr(prhs[2]));
    plhs[0]=mxCreateDoubleMatrix(la-lb+1,1,mxREAL);
    c= mxGetPr(plhs[0]);
    double* d_a,*d_c;
    hipMalloc(&d_a, sizeof(double) * la);
    hipMalloc(&d_c, sizeof(double) * (la-lb+1));
    hipMemcpy(d_a, a, sizeof(double) * la, hipMemcpyHostToDevice);
    checkCUDAError("memcpy");
    // 执行Kernel
    int blocksPerGrid = 256;
    int threadsPerBlock = 256;
    cmax<<<blocksPerGrid, threadsPerBlock>>>(d_a,la,lb,d_c);
    checkCUDAError("corr fail");
    hipMemcpy(c, d_c, sizeof(double) * (la-lb+1), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_c);
    return;
}
